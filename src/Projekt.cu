#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <array>
#include <cmath>
#include <vector>
#include <type_traits>
#include <cassert>
#include <set>
#include <algorithm>
#include <string>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/count.h>

#include ""
#include "CudaUtils.h"

#define PATTERN_LENGTH 4

#define WORK_WIDTH 6 // TODO liczba wariacji
#define WORK_HEIGHT 1
#define BLOCK_WIDTH 1
#define BLOCK_HEIGHT 1
#define WORK_TOTAL WORK_WIDTH * WORK_HEIGHT
#define RESULTS_TOTAL 10
using Type = int;

template <typename T, std::size_t SIZE>
struct VariationOutput {
    bool found;
    T variation[SIZE];
};

template <typename Container>
std::string printContainer(Container const& container) {
    if (container.empty()) {
        return "{}";
    }
    std::string result = "{" + std::to_string(*(container.begin()));
    if (container.size() == 1) {
        return result + "}";
    }
    for (auto it = std::next(container.begin()); it != container.end(); ++it) {
        result += "," + std::to_string(*it);
    }
    result += '}';
    return result;
}


template <typename Integral, typename std::enable_if_t<std::is_integral<Integral>::value>* = nullptr>
__host__ __device__ Integral factorial(Integral const n) {
//    if constexpr (!std::is_unsigned<Integral>::value) {
    assert(n >= 0);
//    }
    if (n == 0) {
        return 1;
    }
    Integral result = 1;
    for (Integral i = 2; i <= n; ++i) {
        result *= i;
    }
    return result;
}

template <typename Integral, typename std::enable_if_t<std::is_integral<Integral>::value>* = nullptr>
__host__ __device__ Integral variationsCount(Integral const n, Integral const k) {
//    if constexpr (!std::is_unsigned<Integral>::value) {
        assert(n >= 0 && k >= 0);
//    }
    return factorial(n) / factorial(n - k);
}

template <typename T, typename Integral, typename std::enable_if_t<std::is_integral<Integral>::value>* = nullptr>
__host__ __device__ void computeVariation(T const* const input, Integral const n, Integral const k, Integral p, T* const output) {
    assert(n >= 0 && k >= 0 && k <= n && p >= 0 && p < variationsCount(n, k));

    // TODO possible optimization - use bitset
    bool* removed = new bool[k];
    for (Integral i = 0; i < k; ++i) {
        removed[i] = false;
    }

    for (Integral x = 0; x <= k - 1; ++x) {
        Integral v = variationsCount(n - x - 1, k - x - 1);
        Integral t = p / v;

        for (Integral i = 0; i <= t; ++i) {
            if (removed[i]) {
                ++t;
            }
        }

        output[x] = input[t];
        removed[t] = true;
        p = p % v;
    }

    delete[] removed;
}

template <typename T, typename S>
__host__ __device__ void substitutePattern(GpuData<T, S> const& pattern, GpuData<T, S> const& distinctPattern,
                                           T const * const variation, T* const output) {

    for (S patternIndex = 0; patternIndex < pattern.length; ++patternIndex) {
        T currentPatternSymbol = pattern.data[patternIndex];
        // Find the substitution
        for (S distinctPatternIndex = 0; distinctPatternIndex < distinctPattern.length; ++distinctPatternIndex) {
            T currentDistinctPatternSymbol = distinctPattern.data[distinctPatternIndex];
            if (currentDistinctPatternSymbol == currentPatternSymbol) {
                output[patternIndex] = variation[distinctPatternIndex];
                break;
            }
        }
    }
}

template <typename T>
void distinctValues(std::vector<T>& data) {
    std::sort(data.begin(), data.end());
    auto iter = std::unique(data.begin(), data.end());
    data.resize(std::distance(data.begin(), iter));
}

template <typename T, typename S>
__host__ __device__ bool checkPattern(GpuData<T, S> const& sequence, GpuData<T, S> const& pattern) {
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;

    T const* sequencePtr = sequence.data;
    T const* patternPtr = pattern.data;
    while (sequencePtr - sequence.data < sequence.length) {
        if (*patternPtr == *sequencePtr) {
            ++patternPtr;
            if (patternPtr - pattern.data == pattern.length) {
                printf("[GTID %d] Matches!\n", gtid);
                return true;
            }
        }
        ++sequencePtr;
    }
    printf("[GTID %d] Not matches!\n", gtid);
    return false;
}

template <typename T, typename S>
__global__ void compute(GpuData<T, S> const sequence, GpuData<T, S> const distinctSequence, GpuData<T, S> const pattern,
                        GpuData<T, S> const distinctPattern, T* outputVariations, bool* outputFound) {

    int const gtid = blockIdx.x * blockDim.x + threadIdx.x;

    T* variation = new T[distinctPattern.length];
    // Compute the variation to be checked by this thread
    computeVariation(distinctSequence.data, distinctSequence.length, distinctPattern.length, gtid, variation);
    T* finalPattern = outputVariations + (gtid * pattern.length);
    // Assign computed values to the pattern
    substitutePattern(pattern, distinctPattern, variation, finalPattern);
    outputFound[gtid] = checkPattern(sequence, GpuData<T, S> { finalPattern, pattern.length });
    // If found a match, copy the substituted pattern to the output array
    if (outputFound[gtid]) {
        for (S i = 0; i < pattern.length; ++i) {
            outputVariations[gtid + i] = finalPattern[i];
        }
    }
    delete[] variation;
}

int main() {

    std::vector<Type> pattern = { 0,1,1,0 };
    CudaBuffer<Type, Type> devPattern(pattern);

    std::vector<Type> sequence = { 1,2,4,3,5,3,6,2,1 };
    CudaBuffer<Type, Type> devSequence(sequence);

    std::vector<Type> distinctPattern(pattern);
    distinctValues(distinctPattern);
    CudaBuffer<Type, Type> devDistinctPattern(distinctPattern);

    std::vector<Type> distinctSequence(sequence);
    distinctValues(distinctSequence);
    CudaBuffer<Type, Type> devDistinctSequence(distinctSequence);

    Type workAmount = variationsCount(distinctSequence.size(), distinctPattern.size());
    CudaBuffer<Type, Type> devOutputVariations(workAmount * sizeof(Type));
//    CudaBuffer<bool, Type> devOutputFound(workAmount);
    thrust::device_vector<bool> devOutputFound(workAmount);

    dim3 dimBlock(workAmount, 1);
    dim3 dimGrid(
            static_cast<int>(ceilf(workAmount / static_cast<float>(dimBlock.x))),
            static_cast<int>(ceilf(1 / static_cast<float>(dimBlock.y)))
    );

    printf("Invoking with: Block(%d,%d), Grid(%d,%d)\n", dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y);
    runWithProfiler([&]{
        compute<Type, Type><<<dimGrid, dimBlock>>>(devSequence, devDistinctSequence, devPattern, devDistinctPattern,
                                                   devOutputVariations, devOutputFound.data().get());
    });
    std::cout << "przed liczeniem" << std::endl;

//    int variationsAmount = thrust::count(devOutputFound.getPointer(), devOutputFound.getPointer() + devOutputFound.getLength() , true);
    auto variationsAmount = thrust::count(devOutputFound.begin(), devOutputFound.end(), true);
    thrust::device_vector<Type> result(variationsAmount);

    auto first = thrust::make_zip_iterator(thrust::make_tuple(devOutputVariations.getPointer(), thrust::counting_iterator<Type>(0)));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(devOutputVariations.getPointer(), thrust::counting_iterator<Type>(workAmount)));
//    thrust::copy_if(devOutputVariations.getPointer(), devOutputVariations.getPointer() + devOutputVariations.getLength(),
//                    __device__ [] (auto v))

    std::vector<Type> outputVariations(devOutputVariations.getLength());
//    std::vector<bool> outputFound(devOutputFound.getLength());

    devOutputVariations.copyTo(outputVariations);
//    devOutputFound.copyTo(outputFound);

    std::cout << printContainer(outputVariations) << std::endl;

}
