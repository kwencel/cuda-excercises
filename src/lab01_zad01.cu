
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

const int N = 1024;
const int blocksize = 16;

__global__ void addMatrix(float *a, float *b, float *c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i + j * N;
    if (i < N && j < N) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    auto* a = new float[N * N];
    auto* b = new float[N * N];
    auto* c = new float[N * N];

    for (int i = 0; i < N * N; ++i) {
        a[i] = 1.0f;
        b[i] = 3.5f;
    }

    float *ad, *bd, *cd;
    const int size = N * N * sizeof(float);
    hipMalloc((void**)&ad, size);
    hipMalloc((void**)&bd, size);
    hipMalloc((void**)&cd, size);

    //    N / bs + ((N % bs) != 0);

    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(blocksize, blocksize);
    dim3 dimGrid((N / dimBlock.x) + 1, (N / dimBlock.y) + 1);
    addMatrix <<<dimGrid, dimBlock >>> (ad, bd, cd, N);

    hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    for (int i = 0; i < N * N; i++) {
        std::cout << i << c[i] << std::endl;
    }

    hipFree(ad);
    hipFree(bd);
    hipFree(cd);
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}
