#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <array>
#include <iostream>
#include "CudaUtils.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define WORK_WIDTH 12
#define WORK_HEIGHT 1
#define BLOCK_WIDTH 12
#define BLOCK_HEIGHT 1
#define WORK_TOTAL WORK_WIDTH * WORK_HEIGHT

template <typename T, size_t SIZE>
__global__ void compute(T* devSrc, T* devRes) {
    __shared__ T sharedSrc[SIZE];
    int tid = threadIdx.x;

    if (tid < SIZE) {
        sharedSrc[tid] = devSrc[tid];
    }
    __syncthreads();
    // Shared memory got filled

    T sum = 0;
    if (tid < SIZE) {
        for (size_t i = 0; i <= tid; ++i) {
            sum += sharedSrc[i];
        }
        devRes[tid] = sum;
    }
}

int main() {
    std::array<int, WORK_TOTAL> src {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
    std::array<int, WORK_TOTAL> res;

    runWithProfiler([&]() {
        CudaBuffer<int> devSrc {src};
        CudaBuffer<int> devRes {WORK_TOTAL};

        dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
        dim3 dimGrid(ceil(WORK_WIDTH / (float) dimBlock.x), ceil(WORK_HEIGHT / (float) dimBlock.y));
        printf("Invoking with: Block(%d,%d), Grid(%d,%d)\n", dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y);
        compute<int, WORK_TOTAL> <<<dimGrid, dimBlock>>> (devSrc, devRes);
        devRes.copyTo(res);
    });

    // Print the results
    for (int col = 0; col < WORK_TOTAL; ++col) {
        std::cout << res[col] << std::endl;
    }
}
