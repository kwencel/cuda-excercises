#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <array>
#include <iostream>
#include "CudaUtils.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include "util/StringConcat.h"
#include <cstdint>

int main() {

    thrust::host_vector<int> src(std::vector<int> { 10, 25, 4, -2, 15, 35, 27, 99, 1 });
    thrust::device_vector<int> devSrc = src;
    thrust::device_vector<uint8_t> devRes(devSrc.size());

    thrust::transform(devSrc.begin(), devSrc.end(), devRes.begin(), [] __device__ (auto v) { return __popc(v); });
    thrust::host_vector<int> res = devRes;
    int sum = thrust::reduce(devRes.begin(), devRes.end(), 0, thrust::plus<int>());

    // Wait for the kernel to complete and check for errors
    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());

    // Print the results
    for (int col = 0; col < res.size(); ++col) {
        std::cout << res[col] << std::endl;
    }
    std::cout << "Sum is: " << sum << std::endl;
}
