#include "hip/hip_runtime.h"
#include ""

#include <array>
#include <iostream>
#include "CudaUtils.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "util/StringConcat.h"
#include <cstdint>

int main() {

    thrust::host_vector<int> src(std::vector<int> { 10, 25, 4, -2, 15, 35, 27, 99, 1 });
    thrust::device_vector<int> devSrc = src;
    thrust::device_vector<std::size_t> devRes(devSrc.size());

    auto isEven = [] __device__ (auto x) { return x % 2 == 0; };
    thrust::transform(devSrc.begin(), devSrc.end(), devRes.begin(), isEven);
    int count = thrust::reduce(devRes.begin(), devRes.end(), 0, thrust::plus<int>());
    thrust::host_vector<int> res = devRes;

    // Wait for the kernel to complete and check for errors
    checkCuda(hipPeekAtLastError());
    checkCuda(hipDeviceSynchronize());

    // Print the results
    for (int col = 0; col < res.size(); ++col) {
        std::cout << res[col] << std::endl;
    }
    std::cout << "Even count is: " << count << std::endl;
}
